#include "hip/hip_runtime.h"
﻿#include <math.h>
#include "Nodal.h"


__global__ void reset(float* d_hmesh, float* d_xstf, float* d_xsdf, float* d_eta1, float* d_eta2, float* d_m260, float* d_m251, float* d_m253, float* d_m262, float* d_m264, float* d_diagD, float* d_diagDI) {
	int lk = threadIdx.x + blockIdx.x * blockDim.x;
	int lkd0 = lk * NDIRMAX;
	int lkg0 = lk * d_ng;

	for (size_t idir = 0; idir < NDIRMAX; idir++)
	{
		int lkd = lkd0 + idir;

		for (size_t ig = 0; ig < d_ng; ig++)
		{
			int lkg = lkg0 + ig;
			auto kp2 = d_xstf[lkg] * d_hmesh(idir, lk) * d_hmesh(idir, lk) / (4 * d_xsdf[lkg]);
			auto kp = sqrt(kp2);
			auto kp3 = kp2 * kp;
			auto kp4 = kp2 * kp2;
			auto kp5 = kp2 * kp3;
			auto rkp = 1 / kp;
			auto rkp2 = rkp * rkp;
			auto rkp3 = rkp2 * rkp;
			auto rkp4 = rkp2 * rkp2;
			auto rkp5 = rkp2 * rkp3;
			auto sinhkp = sinh(kp);
			auto coshkp = cosh(kp);

			//calculate coefficient of basic functions P5and P6
			auto bfcff0 = -sinhkp * rkp;
			auto bfcff2 = -5 * (-3 * kp * coshkp + 3 * sinhkp + kp2 * sinhkp) * rkp3;
			auto bfcff4 = -9. * (-105 * kp * coshkp - 10 * kp3 * coshkp + 105 * sinhkp + 45 * kp2 * sinhkp + kp4 * sinhkp) * rkp5;
			auto bfcff1 = -3 * (kp * coshkp - sinhkp) * rkp2;
			auto bfcff3 = -7 * (15 * kp * coshkp + kp3 * coshkp - 15 * sinhkp - 6 * kp2 * sinhkp) * rkp4;

			auto oddtemp = 1 / (sinhkp + bfcff1 + bfcff3);
			auto eventemp = 1 / (coshkp + bfcff0 + bfcff2 + bfcff4);

			//eta1, eta2
			d_eta1(ig, lkd) = (kp * coshkp + bfcff1 + 6 * bfcff3) * oddtemp;
			d_eta2(ig, lkd) = (kp * sinhkp + 3 * bfcff2 + 10 * bfcff4) * eventemp;

			//set to variables that depends on node properties by integrating of Pi* pj over - 1 ~1
			d_m260(ig, lkd) = 2 * d_eta2(ig, lkd);
			d_m251(ig, lkd) = 2 * (kp * coshkp - sinhkp + 5 * bfcff3) * oddtemp;
			d_m253(ig, lkd) = 2 * (kp * (15 + kp2) * coshkp - 3 * (5 + 2 * kp2) * sinhkp) * oddtemp * rkp2;
			d_m262(ig, lkd) = 2 * (-3 * kp * coshkp + (3 + kp2) * sinhkp + 7 * kp * bfcff4) * eventemp * rkp;
			d_m264(ig, lkd) = 2 * (-5 * kp * (21 + 2 * kp2) * coshkp + (105 + 45 * kp2 + kp4) * sinhkp) * eventemp * rkp3;
			if (d_m264(ig, lkd) == 0.0) d_m264(ig, lkd) = 1.e-10;

			d_diagD(ig, lkd) = 4 * d_xsdf[lkg] / (d_hmesh(idir, lk) * d_hmesh(idir, lk));
			d_diagDI(ig, lkd) = 1.0 / d_diagD(ig, lkd);
		}
	}
}

__global__ void resetMatrix(double& d_reigv, float* d_xstf, float* d_xsnff, float* d_xschif, float* d_xssf, float* d_matMs, float* d_matMf, float* d_matM) {
	int lk = threadIdx.x + blockIdx.x * blockDim.x;
	int lkg0 = lk * d_ng;

	for (size_t igd = 0; igd < d_ng; igd++)
	{
		for (size_t igs = 0; igs < d_ng; igs++)
		{
			d_matMs(igs, igd, lk) = -d_xssf(igs, igd, lk);
			d_matMf(igs, igd, lk) = d_xschif[lkg0 + igd] * d_xsnff[lkg0 + igs];
		}
		d_matMs(igd, igd, lk) += d_xstf[lkg0 + igd];
	}
}

__global__ void prepareMatrix(float* d_m251, float* d_m253, float* d_diagD, float* d_diagDI, float* d_matM, float* d_matMI, float* d_tau, float* d_mu) {
	int lk = threadIdx.x + blockIdx.x * blockDim.x;
	int lkd0 = lk * NDIRMAX;

	auto det = d_matM(0, 0, lk) * d_matM(1, 1, lk) - d_matM(1, 0, lk) * d_matM(0, 1, lk);

	if (abs(det) < 1.E-10) {
		auto rdet = 1 / det;
		d_matMI(0, 0, lk) =  rdet * d_matM(1, 1, lk);
		d_matMI(1, 0, lk) = -rdet * d_matM(1, 0, lk);
		d_matMI(0, 1, lk) = -rdet * d_matM(0, 1, lk);
		d_matMI(1, 1, lk) =  rdet * d_matM(0, 0, lk);
	}
	else {
		d_matMI(0, 0, lk) = 0;
		d_matMI(1, 0, lk) = 0;
		d_matMI(0, 1, lk) = 0;
		d_matMI(1, 1, lk) = 0;
	}

	auto rm011 = 1. / m011;

	for (size_t idir = 0; idir < NDIRMAX; idir++)
	{
		auto lkd = lkd0 + idir;

		float tempz[2][2] = {0.0};

		for (size_t igd = 0; igd < d_ng; igd++)
		{
			auto tau1 = m033 * (d_diagDI(igd, lkd) / d_m253(igd, lkd));

			tempz[igd][igd] = tempz[igd][igd] + m231;

			for (size_t igs = 0; igs < d_ng; igs++)
			{
				d_tau(igs, igd, lkd) = tau1 * d_matM(igs, igd, lk);

				// mu=m011_inv*M_inv*D*(m231*I+m251*tau)
				tempz[igs][igd] += d_m251(igd, lkd) * d_tau(igs, igd, lkd);

				// mu=m011_inv*M_inv*D*(m231*I+m251*tau)
				tempz[igs][igd] *= d_diagD(igd, lkd);
			}
		}

		// mu=m011_inv*M_inv*D*(m231*I+m251*tau)
		d_mu(0, 0, lkd) = rm011 * (d_matMI(0, 0, lk) * tempz[0][0] + d_matMI(1, 0, lk) * tempz[0][1]);
		d_mu(1, 0, lkd) = rm011 * (d_matMI(0, 0, lk) * tempz[1][0] + d_matMI(1, 0, lk) * tempz[1][1]);
		d_mu(0, 1, lkd) = rm011 * (d_matMI(0, 1, lk) * tempz[0][0] + d_matMI(1, 1, lk) * tempz[0][1]);
		d_mu(1, 1, lkd) = rm011 * (d_matMI(0, 1, lk) * tempz[1][0] + d_matMI(1, 1, lk) * tempz[1][1]);


	}

}



__device__ void trlcffbyintg(float* avgtrl3, float* hmesh3, float& trlcff1, float& trlcff2) {
	float sh[4];

	auto rh = (1 / ((hmesh3[1] + hmesh3[0] + hmesh3[2]) * (hmesh3[1] + hmesh3[0]) * (hmesh3[0] + hmesh3[2])));
	sh[1] = (2 * hmesh3[1] + hmesh3[0]) * (hmesh3[1] + hmesh3[0]);
	sh[2] = hmesh3[1] + hmesh3[0];
	sh[3] = (hmesh3[0] + 2 * hmesh3[2]) * (hmesh3[0] + hmesh3[2]);
	sh[4] = hmesh3[0] + hmesh3[2];

	if (hmesh3[LEFT] == 0.0) {
		trlcff1 = 0.125 * (5. * avgtrl3[CENTER] + avgtrl3[RIGHT]);
		trlcff2 = 0.125 * (-3. * avgtrl3[CENTER] + avgtrl3[RIGHT]);
	}
	else if (hmesh3[RIGHT] == 0.0) {
		trlcff1 = -0.125 * (5. * avgtrl3[CENTER] + avgtrl3[LEFT]);
		trlcff2 = 0.125 * (-3. * avgtrl3[CENTER] + avgtrl3[LEFT]);
	}
	else {
		trlcff1 = 0.5 * rh * hmesh3[0] * ((avgtrl3[CENTER] - avgtrl3[LEFT]) * sh[3] + (avgtrl3[RIGHT] - avgtrl3[CENTER]) * sh[1]);
		trlcff2 = 0.5 * rh * (hmesh3[0] * hmesh3[0]) * ((avgtrl3[LEFT] - avgtrl3[CENTER]) * sh[4] + (avgtrl3[RIGHT] - avgtrl3[CENTER]) * sh[2]);
	}

}

__global__ void calculateTransverseLeakage(int* d_lktosfc, int* d_neib, float* d_hmesh, float* d_jnet, float* d_trlcff0, float* d_trlcff1, float* d_trlcff2)
{
	int lk = threadIdx.x + blockIdx.x * blockDim.x;
	int lkd0 = lk * NDIRMAX;

	float avgjnet[NDIRMAX];

	for (size_t ig = 0; ig < d_ng; ig++)
	{
		
		for (size_t idir = 0; idir < NDIRMAX; idir++)
		{
			auto lksl = d_lktosfc(LEFT, idir, lk);
			auto lksr = d_lktosfc(RIGHT, idir, lk);

			avgjnet[idir] = (d_jnet(ig, lksr) - d_jnet(ig, lksl)) * d_hmesh(idir, lk);

			d_trlcff0(ig, lkd0 + XDIR) = avgjnet[YDIR] + avgjnet[ZDIR];
			d_trlcff0(ig, lkd0 + YDIR) = avgjnet[XDIR] + avgjnet[ZDIR];
			d_trlcff0(ig, lkd0 + ZDIR) = avgjnet[XDIR] + avgjnet[YDIR];
		}
	}

	for (size_t idir = 0; idir < NDIRMAX; idir++)
	{
		int lkd = lkd0 + idir;

		float avgtrl3[LRC] = { 0.0 }, hmesh3[LRC] = { 0.0 };

		hmesh3[CENTER] = d_hmesh(idir, lk);

		for (size_t lr = 0; lr < LR; lr++)
		{
			auto lnk = d_neib(lr, idir, lk);
			int lnkd = lnk*NDIRMAX + idir;
			hmesh3[lr] = d_hmesh(idir, lnk);

			for (size_t ig = 0; ig < d_ng; ig++)
			{
				avgtrl3[CENTER] = d_trlcff0(ig, lkd);
				avgtrl3[lr] = d_trlcff0(ig, lnkd);
				trlcffbyintg(avgtrl3, hmesh3, d_trlcff1(ig, lkd), d_trlcff2(ig, lkd));
			}
		}
	}
}

__global__ void calculateEven(float* d_m260, float* d_m262, float* d_m264, float* d_diagD, float* d_diagDI, float* d_matM, double* d_flux, float* d_trlcff0, float* d_trlcff2, float* d_dsncff2, float* d_dsncff4, float* d_dsncff6)
{
	int lk = threadIdx.x + blockIdx.x * blockDim.x;
	int lkd0 = lk * NDIRMAX;

	for (size_t idir = 0; idir < NDIRMAX; idir++)
	{
		auto lkd = lkd0 + idir;
		float at2[2][2], a[2][2], rm4464[2], bt1[2], bt2[2], b[2];

		for (size_t igd = 0; igd < d_ng; igd++)
		{
			rm4464[igd] = m044 / d_m264(igd, lkd);
			auto mu2 = rm4464[igd] * d_m260(igd, lkd) * d_diagDI(igd, lkd);

			for (size_t igs = 0; igs < d_ng; igs++)
			{
				at2[igs][igd] = m022 * rm220 * mu2 * d_matM(igs, igd, lk);
			}
			at2[igd][igd] += m022 * rm220 * m240;
		}

		for (size_t igd = 0; igd < d_ng; igd++)
		{
			auto mu1 = rm4464[igd] * d_m262(igd, lkd);
			for (size_t igs = 0; igs < d_ng; igs++)
			{
				a[igs][igd] = mu1 * d_matM(igs, igd, lk) + d_matM(0, igd, lk) * at2[igs][0] + d_matM(1, igd, lk) * at2[igs][1];
			}
			a[igd][igd] += d_diagD(igd, lkd) * m242;
			bt2[igd] = 2 * (d_matM(0, igd, lk) * d_flux(0, lk) + d_matM(1, igd, lk) * d_flux(0, lk) + d_trlcff0(igd, lkd));
			bt1[igd] = m022 * rm220 * d_diagDI(igd, lkd) * bt2[igd];
		}

		for (size_t ig = 0; ig < d_ng; ig++)
		{
			b[ig] = m220 * d_trlcff2(ig, lkd) + d_matM(0, ig, lk) * bt1[0] + d_matM(1, ig, lk)* bt1[1];
		}

		auto rdet = 1 / (a[0][0] * a[1][1] - a[1][0] * a[0][1]);
		d_dsncff4(0, lkd) = rdet * (a[1][1] * b[0] - a[1][0] * b[1]);
		d_dsncff4(1, lkd) = rdet * (a[0][0] * b[1] - a[0][1] * b[0]);

		for (size_t ig = 0; ig < d_ng; ig++)
		{
			d_dsncff6(ig, lkd) = d_diagDI(ig, lkd) * rm4464[ig] * (d_matM(0, ig, lk) * d_dsncff4(0, lkd) + d_matM(1, ig, lk) * d_dsncff4(1, lkd));
			d_dsncff2(ig, lkd) = rm220 * (d_diagDI(ig, lkd) * bt2[ig] - m240 * d_dsncff4(ig, lkd) - d_m260(ig, lkd) * d_dsncff6(ig, lkd));
		}
	}
}

__global__ void calculateJnet(int * lklr, int * idirlr, int* sgnlr,float * d_hmesh, float* d_xsadf, float* d_m260, float* d_m262, float* d_m264, float* d_diagD, float* d_diagDI, float* d_matM, float* d_matMI, double* d_flux, float* d_trlcff0, float* d_trlcff1, float* d_trlcff2, float* d_mu, float* d_tau, float* d_eta1, float* d_eta2, float* d_dsncff2, float* d_dsncff4, float* d_dsncff6, float* d_jnet)
{
	int lsfc = threadIdx.x + blockIdx.x * blockDim.x;
	int lsfclr = lsfc * LR;

	int lkl		= lklr[lsfclr + LEFT];
	int lkr		= lklr[lsfclr + RIGHT];
	int idirl = idirlr[lsfclr + LEFT];
	int idirr = idirlr[lsfclr + RIGHT];
	int sgnl = sgnlr[lsfclr+ LEFT];
	int sgnr = sgnlr[lsfclr + RIGHT];
	int lkdl = lkl * NDIRMAX + idirl;
	int lkdr = lkr * NDIRMAX + idirr;

	float adf[2][LR], d_diagDj[2][LR], tempz[2][2], tempzI[2][2], zeta1[2][2], zeta2[2], bfc[2], mat1g[2][2];

	for (size_t ig = 0; ig < d_ng; ig++)
	{
		adf[ig][LEFT] = d_xsadf(ig, lkl);
		adf[ig][RIGHT] = d_xsadf(ig, lkr);
		d_diagDj[ig][LEFT] =  0.5 * d_hmesh(idirl, lkl) * d_diagD(ig, lkdl);
		d_diagDj[ig][RIGHT] = 0.5 * d_hmesh(idirr,lkr) * d_diagD(ig, lkdr);
	}

	//zeta1 = (d_mur + I + d_taur)_inv * (d_mul + I + d_taul)
	tempz[0][0] = (d_mu(0, 0, lkdr) + d_tau(0, 0, lkdr) + 1) * adf[0][RIGHT];
	tempz[1][0] = (d_mu(1, 0, lkdr) + d_tau(1, 0, lkdr)) * adf[0][RIGHT];
	tempz[0][1] = (d_mu(0, 1, lkdr) + d_tau(0, 1, lkdr)) * adf[1][RIGHT];
	tempz[1][1] = (d_mu(1, 1, lkdr) + d_tau(1, 1, lkdr) + 1) * adf[1][RIGHT];

	auto rdet = 1 / (tempz[0][0] * tempz[1][1] - tempz[1][0] * tempz[0][1]);
	tempzI[0][0] = rdet * tempz[1][1];
	tempzI[1][0] = -rdet * tempz[1][0];
	tempzI[0][1] = -rdet * tempz[0][1];
	tempzI[1][1] = rdet * tempz[0][0];

	tempz[0][0] = (d_mu(0, 0, lkdl) + d_tau(0, 0, lkdl) + 1) * adf[0][LEFT];
	tempz[1][0] = (d_mu(1, 0, lkdl) + d_tau(1, 0, lkdl)) * adf[0][LEFT];
	tempz[0][1] = (d_mu(0, 1, lkdl) + d_tau(0, 1, lkdl)) * adf[1][LEFT];
	tempz[1][1] = (d_mu(1, 1, lkdl) + d_tau(1, 1, lkdl) + 1) * adf[1][LEFT];

	zeta1[0][0] = tempzI[0][0] * tempz[0][0] + tempzI[1][0] * tempz[0][1];
	zeta1[1][0] = tempzI[0][0] * tempz[1][0] + tempzI[1][0] * tempz[1][1];
	zeta1[0][1] = tempzI[0][1] * tempz[0][0] + tempzI[1][1] * tempz[0][1];
	zeta1[1][1] = tempzI[0][1] * tempz[1][0] + tempzI[1][1] * tempz[1][1];

	for (size_t ig = 0; ig < d_ng; ig++)
	{
		bfc[ig] = adf[ig][RIGHT] * (d_dsncff2(ig, lkdr) + d_dsncff4(ig, lkdr) + d_dsncff6(ig, lkdr)
					+ d_flux(ig, lkr) + d_matMI(0, ig, lkr) * sgnr * d_trlcff1(0, lkdr)
					+ d_matMI(1, ig, lkr) * sgnr * d_trlcff1(1, lkdr))
				+ adf[ig][LEFT] * (-d_dsncff2(ig, lkdl) - d_dsncff4(ig, lkdl) - d_dsncff6(ig, lkdl)
					- d_flux(ig, lkl) + d_matMI(0, ig, lkl) * sgnl * d_trlcff1(0, lkdl)
					+ d_matMI(1, ig, lkl) * sgnl * d_trlcff1(1, lkdl));
	}

	for (size_t ig = 0; ig < d_ng; ig++)
	{
		zeta2[ig] = tempzI[0][ig] * bfc[0] + tempzI[1][ig] * bfc[1];
	}

	//tempz = d_mur + 6 * I + eta1 * d_taur
	tempz[0][0] = d_diagDj[0][RIGHT] * (d_mu(0, 0, lkdr) + 6 + d_eta1(0, lkdr) * d_tau(0, 0, lkdr));
	tempz[1][0] = d_diagDj[0][RIGHT] * (d_mu(1, 0, lkdr) + d_eta1(0, lkdr) * d_tau(1, 0, lkdr));
	tempz[0][1] = d_diagDj[1][RIGHT] * (d_mu(0, 1, lkdr) + d_eta1(1, lkdr) * d_tau(0, 1, lkdr));
	tempz[1][1] = d_diagDj[1][RIGHT] * (d_mu(1, 1, lkdr) + 6 + d_eta1(1, lkdr) * d_tau(1, 1, lkdr));


	//mat1g = d_mul + 6 * I + eta1 * d_taul - tempzI
	mat1g[0][0] = -d_diagDj[0][LEFT] * (d_mu(0, 0, lkdl) + 6 + d_eta1(0, lkdl) * d_tau(0, 0, lkdl)) - tempz[0][0] * zeta1[0][0] - tempz[1][0] * zeta1[0][1];
	mat1g[1][0] = -d_diagDj[0][LEFT] * (d_mu(1, 0, lkdl) + d_eta1(0, lkdl) * d_tau(1, 0, lkdl)) - tempz[0][0] * zeta1[1][0] - tempz[1][0] * zeta1[1][1];
	mat1g[0][1] = -d_diagDj[1][LEFT] * (d_mu(0, 1, lkdl) + d_eta1(1, lkdl) * d_tau(0, 1, lkdl)) - tempz[0][1] * zeta1[0][0] - tempz[1][1] * zeta1[0][1];
	mat1g[1][1] = -d_diagDj[1][LEFT] * (d_mu(1, 1, lkdl) + 6 + d_eta1(1, lkdl) * d_tau(1, 1, lkdl)) - tempz[0][1] * zeta1[1][0] - tempz[1][1] * zeta1[1][1];


	float bcc[2], vec1g[2];
		
	for (size_t ig = 0; ig < d_ng; ig++)
	{
		bcc[ig] = d_diagDj[ig][LEFT] * (3 * d_dsncff2(ig, lkdl) + 10 * d_dsncff4(ig, lkdl) + d_eta2(ig, lkdl) * d_dsncff6(ig, lkdl))
			+ d_diagDj[ig][RIGHT] * (3 * d_dsncff2(ig, lkdr) + 10 * d_dsncff4(ig, lkdr) + d_eta2(ig, lkdr) * d_dsncff6(ig, lkdr));
		vec1g[ig] = bcc[ig]
			- d_diagDj[ig][LEFT] * (d_matMI(0, ig, lkl) * sgnl * d_trlcff1(0, lkdl) + d_matMI(0, ig, lkl) * sgnl * d_trlcff1(0, lkdl))
			+ d_diagDj[ig][RIGHT] * (d_matMI(1, ig, lkr) * sgnr * d_trlcff1(1, lkdr) + d_matMI(1, ig, lkr) * sgnr * d_trlcff1(1, lkdr))
			- (tempz[0][ig] * zeta2[0] + tempz[1][ig] * zeta2[1]);

	}

	rdet = 1 / (mat1g[0][0] * mat1g[1][1] - mat1g[1][0] * mat1g[0][1]);
	auto tmp = mat1g[0][0];
	mat1g[0][0] = rdet * mat1g[1][1];
	mat1g[1][0] = -rdet * mat1g[1][0];
	mat1g[0][1] = -rdet * mat1g[0][1];
	mat1g[1][1] = rdet * tmp;

	float oddcff[3][2];

	oddcff[1][0] = zeta2[0] - (zeta1[0][0] * (mat1g[0][0] * vec1g[0] + mat1g[1][0] * vec1g[1])
		+ zeta1[1][0] * (mat1g[0][1] * vec1g[0] + mat1g[1][1] * vec1g[1]));
	oddcff[1][1] = zeta2[1] - (zeta1[0][1] * (mat1g[0][0] * vec1g[0] + mat1g[1][0] * vec1g[1]) 
		+ zeta1[1][1] * (mat1g[0][1] * vec1g[0] + mat1g[1][1] * vec1g[1]));

	oddcff[2][0] = d_tau(0, 0, lkdr) * oddcff[1][0] + d_tau(1, 0, lkdr) * oddcff[1][1];
	oddcff[2][1] = d_tau(0, 1, lkdr) * oddcff[1][0] + d_tau(1, 1, lkdr) * oddcff[1][1];

	oddcff[0][0] = d_mu(0, 0, lkdr) * oddcff[1][0] - d_matMI(0, 0, lkr) * sgnr * d_trlcff1(0, lkdr)
		+ d_mu(1, 0, lkdr) * oddcff[1][1] - d_matMI(1, 0, lkr) * sgnr * d_trlcff1(1, lkdr);
	oddcff[0][1] = d_mu(0, 1, lkdr) * oddcff[1][0] - d_matMI(0, 1, lkr) * sgnr * d_trlcff1(0, lkdr) 
		+ d_mu(1, 1, lkdr) * oddcff[1][1] - d_matMI(1, 1, lkr) * sgnr * d_trlcff1(1, lkdr);

	for (size_t ig = 0; ig < d_ng; ig++)
	{
		d_jnet(ig, lsfc) = sgnr * d_hmesh(idirr, lkr) * 0.5 * d_diagD(ig, lkdr) * (
			-1.0 * oddcff[0][ig] + 3 * d_dsncff2(ig, lkdr) - 6 * oddcff[1][ig] + 10 * d_dsncff4(ig, lkdr)
			- d_eta1(ig, lkdr) * oddcff[2][ig] + d_eta2(ig, lkdr) * d_dsncff6(ig, lkdr));
	}

}

Nodal::~Nodal()
{
}

Nodal::Nodal()
{
}

void Nodal::init() {
	_blocks = dim3(d_nxyz / NTHREADSPERBLOCK + 1, 1, 1);
	_threads = dim3(NTHREADSPERBLOCK, 1, 1);

	_blocks_sfc = dim3(d_nsurf / NTHREADSPERBLOCK + 1, 1, 1);
	_threads_sfc = dim3(NTHREADSPERBLOCK, 1, 1);

}

void Nodal::drive()
{
	::reset << <_blocks, _threads >> > (d_hmesh, d_xstf, d_xsdf, d_eta1, d_eta2, d_m260, d_m251, d_m253, d_m262, d_m264, d_diagD, d_diagDI);
	::calculateTransverseLeakage << <_blocks, _threads >> > (d_lktosfc, d_neib, d_hmesh, d_jnet, d_trlcff0,  d_trlcff1, d_trlcff2);
	::resetMatrix << <_blocks, _threads >> > (*d_reigv, d_xstf, d_xsnff, d_xschif, d_xssf, d_matMs, d_matMf, d_matM);
	::prepareMatrix << <_blocks, _threads >> > (d_m251, d_m253, d_diagD, d_diagDI, d_matM, d_matMI, d_tau, d_mu);

	::calculateEven << <_blocks, _threads >> > (d_m260, d_m262, d_m264, d_diagD, d_diagDI, d_matM, d_flux,
		d_trlcff0, d_trlcff2, d_dsncff2, d_dsncff4, d_dsncff6);
	::calculateJnet << <_blocks_sfc, _threads_sfc >> > (d_lklr, d_idirlr, d_sgnlr, d_hmesh, d_xsadf, d_m260, d_m262, d_m264,
		d_diagD, d_diagDI, d_matM, d_matMI, d_flux, d_trlcff0, d_trlcff1,
		d_trlcff2, d_mu, d_tau, d_eta1, d_eta2, d_dsncff2, d_dsncff4, d_dsncff6, d_jnet);
}

